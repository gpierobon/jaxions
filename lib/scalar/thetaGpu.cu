#include "hip/hip_runtime.h"
#include "kernelParms.cuh"
#include "complexGpu.cuh"
#include "utils/index.cuh"

#include "enum-field.h"
#include "scalar/scalarField.h"
//#include "utils/utils.h"

using namespace gpuCu;
using namespace indexHelper;

template<class Float>
static __device__ __forceinline__ void	toThetaCoreGpu (const uint idx, const uint cIdx, const uint bIdx, complex<Float> *mC, Float *m, complex<Float> *vC, Float *v, Float z, const uint S, const Float shift)
{
  complex<Float> mTmp = mC[cIdx] - complex<Float>(shift,0.);

	Float iMod = z/(mTmp.real()*mTmp.real() + mTmp.imag()*mTmp.imag());
	m[idx]	   = arg(mTmp);
	m[bIdx]	   = (vC[cIdx-S]*conj(mTmp)).imag()*iMod + m[idx];
	m[idx]	  *= z;
}

template<typename Float>
__global__ void toThetaKernelGpu (complex<Float> *mC, Float *m, complex<Float> *vC, Float *v, Float z, const uint S, const uint ofC, const uint ofB, const Float shift)
{
	const uint idx = (threadIdx.x + blockDim.x*(blockIdx.x + gridDim.x*blockIdx.y));

	if      (idx >= S)
		return;

	const uint cIdx = idx + ofC;
	const uint bIdx = idx + ofB;

	toThetaCoreGpu (idx, cIdx, bIdx, mC, m, vC, v, z, S, shift);
}

template<typename Float>
void	toThetaTemplateGpu (Scalar *sField, const Float shift)
{
	const uint V  = sField->Size();
	const uint S  = sField->Surf();
	const uint Lz = sField->Depth();
	const uint Lx = sField->Length();
	const uint Go = 2*(V+S);

	#define BSSIZE 512
	dim3 gridSize((Lx*Lx+BSSIZE-1)/BSSIZE,1,1);
	dim3 blockSize(BSSIZE,1,1);

	Float *m  = static_cast<Float*>(sField->mGpu());
	Float *v  = static_cast<Float*>(sField->mGpu()) + 2*S + V;
	Float *vT = static_cast<Float*>(sField->vGpu());

	complex<Float> *mC = static_cast<complex<Float>*>(sField->mGpu());
	complex<Float> *vC = static_cast<complex<Float>*>(sField->vGpu());

	const Float z = (Float) (*sField->zV());

	for (uint cZ = 1; cZ < Lz+1; cZ++)
	{
		const uint Vo = cZ*S;

		toThetaKernelGpu<Float><<<gridSize,blockSize,0,((hipStream_t *)sField->Streams())[0]>>>(mC, m, vC, v, z, S, Vo, Go, shift);

		hipMemcpy (m + Vo,      m,      sizeof(Float)*S, hipMemcpyDeviceToDevice);
		hipMemcpy (vT + Vo - S, m + Go, sizeof(Float)*S, hipMemcpyDeviceToDevice);
	}

	hipMemcpy (v, vT, sizeof(Float)*V, hipMemcpyDeviceToDevice);
}

void	toThetaGpu (Scalar *sField, const double shift)
{
	switch (sField->Precision())
	{
		case FIELD_DOUBLE:

			toThetaTemplateGpu<double> (sField, shift);
			break;

		case FIELD_SINGLE:

			toThetaTemplateGpu<float>  (sField, (float) shift);
			break;

		default:

			LogError ("Wrong precision");
			exit  (1);
			break;
	}
}
