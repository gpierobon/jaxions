#include "hip/hip_runtime.h"
#include "complexGpu.cuh"
#include "utils/index.cuh"

#include "enum-field.h"

#include "scalar/varNQCD.h"
#include "utils/parse.h"

#include "utils/reduceGpu.cuh"

#define	BLSIZE 512

using namespace gpuCu;
using namespace indexHelper;

template<const VqcdType VQcd, typename Float>
static __device__ __forceinline__ void	energyCoreGpu(const uint idx, const complex<Float> * __restrict__ m, const complex<Float> * __restrict__ v, const uint Lx, const uint Sf, const double iZ, const double iZ2, double *tR, const Float shift)
{
	uint X[3], idxPx, idxPy, idxMx, idxMy;

	complex<Float> mPX, mPY, mPZ, mMX, mMY, mMZ, tmp, tp2, vOm;

	idx2Vec(idx, X, Lx);

	if (X[0] == Lx-1)
		idxPx = idx - Lx+1;
	else
		idxPx = idx+1;

	if (X[0] == 0)
		idxMx = idx + Lx-1;
	else
		idxMx = idx-1;

	if (X[1] == Lx-1)
		idxPy = idx - Sf + Lx;
	else
		idxPy = idx + Lx;

	if (X[1] == 0)
		idxMy = idx + Sf - Lx;
	else
		idxMy = idx - Lx;

	tmp = m[idx];
	tp2 = tmp - shift;

	Float mod = tmp.real()*tmp.real() + tmp.imag()*tmp.imag();
	Float md2 = tp2.real()*tp2.real() + tp2.imag()*tp2.imag();
	Float mFac = iZ2*mod;
	Float mFc2 = iZ2*md2;
	Float iMod = 1./mod;

	mPX = (m[idxPx]  - tmp)*conj(tmp)*iMod;
	mPY = (m[idxPy]  - tmp)*conj(tmp)*iMod;
	mPZ = (m[idx+Sf] - tmp)*conj(tmp)*iMod;
	mMX = (m[idxMx]  - tmp)*conj(tmp)*iMod;
	mMY = (m[idxMy]  - tmp)*conj(tmp)*iMod;
	mMZ = (m[idx-Sf] - tmp)*conj(tmp)*iMod;
	vOm = v[idx-Sf]*conj(tmp)*iMod - gpuCu::complex<Float>(iZ, 0.);

	tR[RH_GRX] = (double) ((Float) (mFac*(mPX.real()*mPX.real() + mMX.real()*mMX.real())));
	tR[TH_GRX] = (double) ((Float) (mFac*(mPX.imag()*mPX.imag() + mMX.imag()*mMX.imag())));
	tR[RH_GRY] = (double) ((Float) (mFac*(mPY.real()*mPY.real() + mMY.real()*mMY.real())));
	tR[TH_GRY] = (double) ((Float) (mFac*(mPY.imag()*mPY.imag() + mMY.imag()*mMY.imag())));
	tR[RH_GRZ] = (double) ((Float) (mFac*(mPZ.real()*mPZ.real() + mMZ.real()*mMZ.real())));
	tR[TH_GRZ] = (double) ((Float) (mFac*(mPZ.imag()*mPZ.imag() + mMZ.imag()*mMZ.imag())));
	tR[RH_POT] = (double) ((Float) (mFc2 - 1.)*(mFc2 - 1.));
	tR[RH_KIN] = (double) ((Float) (mFac*vOm.real()*vOm.real()));
	tR[TH_KIN] = (double) ((Float) (mFac*vOm.imag()*vOm.imag()));

	switch (VQcd) {
		case	VQCD_1:
			tR[TH_POT] = (double) (((Float) 1.) - tp2.real()/sqrt(md2));
			break;
		case	VQCD_2:
			double smp = (double) (((Float) 1.) - tp2.real()*iZ);
			tR[TH_POT] = smp*smp;
	}
}

template<const VqcdType VQcd, typename Float>
__global__ void	energyKernel(const complex<Float> * __restrict__ m, const complex<Float> * __restrict__ v, const uint Lx, const uint Sf, const uint V, const double iZ, const double iZ2, double *eR, double *partial, const Float shift)
{
	uint idx = Sf + (threadIdx.x + blockDim.x*(blockIdx.x + gridDim.x*blockIdx.y));

	double tmp[10] = { 0., 0., 0., 0., 0., 0., 0., 0., 0., 0. };

	if	(idx < V)
		energyCoreGpu<VQcd,Float>(idx, m, v, Lx, Sf, iZ, iZ2, tmp, shift);

	reduction<BLSIZE,double,10>   (eR, tmp, partial);
}

int	energyGpu	(const void * __restrict__ m, const void * __restrict__ v, double *z, const double delta2, const double LL, const double nQcd, const double shift,
			 const VqcdType VQcd, const uint Lx, const uint Lz, const uint V, const uint S, FieldPrecision precision, double *eR, hipStream_t &stream, const bool map)
{
	const uint Vm = V+S;
	const uint Lz2 = V/(Lx*Lx);
	dim3  gridSize((Lx*Lx+BLSIZE-1)/BLSIZE,Lz2,1);
	dim3  blockSize(BLSIZE,1,1);
	const int nBlocks = gridSize.x*gridSize.y;

	const double zR   = *z;
	double *tR, *partial;

	if ((hipMalloc(&tR, sizeof(double)*10) != hipSuccess) || (hipMalloc(&partial, sizeof(double)*10*nBlocks*4) != hipSuccess))
	{
		return -1;
	}

	if (precision == FIELD_DOUBLE)
	{
		const double iZ  = 1./zR;
		const double iZ2 = iZ*iZ;

		switch (VQcd) {
			case	VQCD_1:
				energyKernel<VQCD_1><<<gridSize,blockSize,0,stream>>> (static_cast<const complex<double>*>(m), static_cast<const complex<double>*>(v), Lx, S, Vm, iZ, iZ2, tR, partial, shift);
				break;

			case	VQCD_2:
				energyKernel<VQCD_2><<<gridSize,blockSize,0,stream>>> (static_cast<const complex<double>*>(m), static_cast<const complex<double>*>(v), Lx, S, Vm, iZ, iZ2, tR, partial, shift);
				break;
		}
	}
	else if (precision == FIELD_SINGLE)
	{
		const float iZ = 1./zR;
		const float iZ2 = iZ*iZ;

		switch (VQcd) {
			case	VQCD_1:
				energyKernel<VQCD_1><<<gridSize,blockSize,0,stream>>> (static_cast<const complex<float>*>(m), static_cast<const complex<float>*>(v), Lx, S, Vm, iZ, iZ2, tR, partial, (float) shift);
				break;

			case	VQCD_2:
				energyKernel<VQCD_2><<<gridSize,blockSize,0,stream>>> (static_cast<const complex<float>*>(m), static_cast<const complex<float>*>(v), Lx, S, Vm, iZ, iZ2, tR, partial, (float) shift);
				break;
		}
	}

	hipDeviceSynchronize();

	hipMemcpy(eR, tR, sizeof(double)*10, hipMemcpyDeviceToHost);
	hipFree(tR); hipFree(partial);

	const double o2 = 0.25/delta2;
	const double zQ = axionmass2(zR, nQcd, zthres, zrestore)*zR*zR;//9.*pow(zR, nQcd+2.);
	const double lZ = 0.25*LL*zR*zR;

	eR[TH_GRX] *= o2;
	eR[TH_GRY] *= o2;
	eR[TH_GRZ] *= o2;
	eR[TH_KIN] *= .5;
	eR[TH_POT] *= zQ;
	eR[RH_GRX] *= o2;
	eR[RH_GRY] *= o2;
	eR[RH_GRZ] *= o2;
	eR[RH_KIN] *= .5;
	eR[RH_POT] *= lZ;

	return	0;
}
